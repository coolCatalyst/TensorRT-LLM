#include "hip/hip_runtime.h"
/*
 * SPDX-FileCopyrightText: Copyright (c) 1993-2022 NVIDIA CORPORATION &
 * AFFILIATES. All rights reserved. SPDX-License-Identifier: Apache-2.0
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/memoryUtils.h"
#include "tensorrt_llm/runtime/runtimeKernels.h"

#include <hipcub/hipcub.hpp>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>

using namespace tensorrt_llm::runtime;
namespace tc = tensorrt_llm::common;

namespace tensorrt_llm::runtime::kernels
{

namespace
{

template <typename T>
__global__ void fill(T* data, std::size_t size, T const value)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;

    for (auto idx = tidx; idx < size; idx += stride)
    {
        data[idx] = value;
    }
}
} // namespace

template <typename T>
void invokeFill(IBuffer& buffer, T const value, CudaStream const& stream)
{
    auto data = bufferCast<T>(buffer);
    auto const size = buffer.getSize();
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax))};

    fill<<<gridSize, blockSize, 0, stream.get()>>>(data, size, value);
}

// template instantiation
template void invokeFill(IBuffer&, std::int32_t, CudaStream const&);
template void invokeFill(IBuffer&, std::int8_t, CudaStream const&);
template void invokeFill(IBuffer&, float, CudaStream const&);

namespace
{
template <typename T>
__global__ void add(T* data, std::size_t size, T const value)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const stride = static_cast<std::size_t>(blockDim.x) * gridDim.x;

    for (auto idx = tidx; idx < size; idx += stride)
    {
        data[idx] += value;
    }
}
} // namespace

template <typename T>
void invokeAdd(IBuffer& buffer, T const value, CudaStream const& stream)
{
    auto data = bufferCast<T>(buffer);
    auto const size = buffer.getSize();
    dim3 const blockSize{256};
    std::size_t const gridx{tc::ceilDiv(size, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax))};

    add<<<gridSize, blockSize, 0, stream.get()>>>(data, size, value);
}

template void invokeAdd(IBuffer&, std::int32_t, CudaStream const&);
template void invokeAdd(IBuffer&, std::int8_t, CudaStream const&);
template void invokeAdd(IBuffer&, float, CudaStream const&);

namespace
{
__global__ void transpose(SizeType* output, SizeType const* input, SizeType const batchSize, SizeType const rowSize)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        for (SizeType tokenIdx = tidx; tokenIdx < rowSize; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const inputIdx = batchIdx * rowSize + tokenIdx;
            auto const outputIdx = tokenIdx * batchSize + batchIdx;
            output[outputIdx] = input[inputIdx];
        }
    }
}
} // namespace

void invokeTranspose(ITensor& output, ITensor const& input, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(input.getDataType() == output.getDataType(), "Input and output have different data types");
    TLLM_CHECK_WITH_INFO(input.getSize() == output.getSize(),
        common::fmtstr("Input size (%ld) and output size (%ld) differ", input.getSize(), output.getSize()));

    auto const& inputShape = input.getShape();
    TLLM_CHECK_WITH_INFO(
        inputShape.nbDims == 2, common::fmtstr("Input shape must have 2 dimensions, but has %d", inputShape.nbDims));

    SizeType const batchSize = inputShape.d[0];
    SizeType const rowSize = inputShape.d[1];

    dim3 const blockSize(256, 1);
    dim3 const gridSize((rowSize + blockSize.x - 1) / blockSize.x, batchSize);

    transpose<<<gridSize, blockSize, 0, stream.get()>>>(
        bufferCast<SizeType>(output), bufferCast<SizeType const>(input), batchSize, rowSize);
}

namespace
{
__global__ void transposeWithOutputOffset(SizeType* output, SizeType const* input, SizeType const nbInputRows,
    SizeType const inputRowSize, SizeType const outputRowSize, SizeType const outputOffset)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < nbInputRows; batchIdx += blockDim.y * gridDim.y)
    {
        for (SizeType tokenIdx = tidx; tokenIdx < inputRowSize; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const inputIdx = batchIdx * inputRowSize + tokenIdx;
            auto const outputIdx = tokenIdx * outputRowSize + outputOffset + batchIdx;
            output[outputIdx] = input[inputIdx];
        }
    }
}
} // namespace

void invokeTransposeWithOutputOffset(
    ITensor& output, ITensor const& input, SizeType const outputOffset, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(input.getDataType() == output.getDataType(), "Input and output have different data types");

    auto const& inputShape = input.getShape();
    TLLM_CHECK_WITH_INFO(
        inputShape.nbDims == 2, common::fmtstr("Input shape must have 2 dimensions, but has %d", inputShape.nbDims));
    SizeType const nbInputRows = inputShape.d[0];
    SizeType const inputRowSize = inputShape.d[1];

    auto const& outputShape = output.getShape();
    TLLM_CHECK_WITH_INFO(
        outputShape.nbDims == 2, common::fmtstr("Output shape must have 2 dimensions, but has %d", outputShape.nbDims));
    SizeType const nbOutputRows = outputShape.d[0];
    SizeType const outputRowSize = outputShape.d[1];

    TLLM_CHECK_WITH_INFO(inputRowSize == nbOutputRows,
        common::fmtstr("Input dim 1 (%d) and output dim 0 (%d) differ", inputRowSize, nbOutputRows));
    TLLM_CHECK_WITH_INFO(outputOffset + nbInputRows <= outputRowSize,
        common::fmtstr("Input (%d rows) does not fit into output (%d columns, offset %d)", nbInputRows, inputRowSize,
            outputOffset));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((inputRowSize + blockSize.x - 1) / blockSize.x, nbInputRows);

    transposeWithOutputOffset<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(output),
        bufferCast<SizeType const>(input), nbInputRows, inputRowSize, outputRowSize, outputOffset);
}

namespace
{
__global__ void transposeWithInputOffset(SizeType* output, SizeType const* input, SizeType const outputRowSize,
    SizeType const nbOutputRows, SizeType const inputRowSize, SizeType const inputOffset)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < outputRowSize; batchIdx += blockDim.y * gridDim.y)
    {
        for (SizeType tokenIdx = tidx; tokenIdx < nbOutputRows; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const inputIdx = batchIdx * inputRowSize + inputOffset + tokenIdx;
            auto const outputIdx = tokenIdx * outputRowSize + batchIdx;
            output[outputIdx] = input[inputIdx];
        }
    }
}
} // namespace

void invokeTransposeWithInputOffset(
    ITensor& output, ITensor const& input, SizeType const inputOffset, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(input.getDataType() == output.getDataType(), "Input and output have different data types");

    auto const& inputShape = input.getShape();
    TLLM_CHECK_WITH_INFO(
        inputShape.nbDims == 2, common::fmtstr("Input shape must have 2 dimensions, but has %d", inputShape.nbDims));
    SizeType const nbInputRows = inputShape.d[0];
    SizeType const inputRowSize = inputShape.d[1];

    auto const& outputShape = output.getShape();
    TLLM_CHECK_WITH_INFO(
        outputShape.nbDims == 2, common::fmtstr("Output shape must have 2 dimensions, but has %d", outputShape.nbDims));
    SizeType const nbOutputRows = outputShape.d[0];
    SizeType const outputRowSize = outputShape.d[1];

    TLLM_CHECK_WITH_INFO(nbInputRows == outputRowSize,
        common::fmtstr("Input dim 0 (%d) and output dim 1 (%d) differ", nbInputRows, outputRowSize));
    TLLM_CHECK_WITH_INFO(inputOffset + nbOutputRows <= inputRowSize,
        common::fmtstr("Cannot extract output (%d rows) from input (%d columns, offset %d)", nbOutputRows, inputRowSize,
            inputOffset));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((nbOutputRows + blockSize.x - 1) / blockSize.x, outputRowSize);

    transposeWithInputOffset<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(output),
        bufferCast<SizeType const>(input), outputRowSize, nbOutputRows, inputRowSize, inputOffset);
}

void invokeInclusiveSum(IBuffer& output, IBuffer const& input, BufferManager const& manager, CudaStream const& stream)
{
    auto const size = input.getSize();
    auto const* inputData = bufferCast<SizeType>(input);
    auto* outputData = bufferCast<SizeType>(output);

    std::size_t tempStorageBytes{0};
    hipcub::DeviceScan::InclusiveSum(nullptr, tempStorageBytes, inputData, outputData, size, stream.get());
    auto tempStorage = manager.gpu(tempStorageBytes, nvinfer1::DataType::kUINT8);
    auto* tempStorageData = bufferCast<std::uint8_t>(*tempStorage);
    hipcub::DeviceScan::InclusiveSum(tempStorageData, tempStorageBytes, inputData, outputData, size, stream.get());
}

namespace
{
__global__ void buildTokenMask(SizeType* tokenMask, SizeType const* inputLengths, SizeType const batchSize,
    SizeType const maxInputLength, SizeType const maxSeqLength)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const inputLength = inputLengths[batchIdx];
        for (SizeType tokenIdx = tidx; tokenIdx < maxSeqLength; tokenIdx += blockDim.x * gridDim.x)
        {
            tokenMask[batchIdx * maxSeqLength + tokenIdx]
                = (tokenIdx >= inputLength && tokenIdx < maxInputLength) ? 1 : 0;
        }
    }
}
} // namespace

void invokeBuildTokenMask(
    ITensor& tokenMask, ITensor const& inputLengths, SizeType const maxInputLength, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(TRTDataType<SizeType>::value == tokenMask.getDataType(), "tokenMask has wrong data type");
    TLLM_CHECK_WITH_INFO(
        TRTDataType<SizeType>::value == inputLengths.getDataType(), "inputLengths has wrong data type");

    auto const& shape = tokenMask.getShape();
    SizeType const batchSize = shape.d[0];
    SizeType const maxSeqLength = shape.d[1];

    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "TtokenMask dimension 1 (%d) is smaller than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxSeqLength + blockSize.x - 1) / blockSize.x, batchSize);

    buildTokenMask<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(tokenMask),
        bufferCast<SizeType const>(inputLengths), batchSize, maxInputLength, maxSeqLength);
}

namespace
{
__global__ void buildAttentionMask(SizeType* attentionMask, SizeType const size, SizeType const padId)
{
    SizeType const tid = blockIdx.x * blockDim.x + threadIdx.x;

    for (SizeType i = tid; i < size; i += blockDim.x * gridDim.x)
    {
        auto const x = attentionMask[i];
        attentionMask[i] = (x != padId);
    }
}
} // namespace

void invokeBuildAttentionMask(ITensor& attentionMask, SizeType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        TRTDataType<SizeType>::value == attentionMask.getDataType(), "attentionMask has wrong data type");

    auto const size = attentionMask.getSize();
    dim3 const blockSize(256);
    dim3 const gridSize((size + blockSize.x - 1) / blockSize.x);

    buildAttentionMask<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(attentionMask), size, padId);
}

namespace
{
__global__ void extendAttentionMask(
    SizeType* newMask, SizeType const* oldMask, SizeType const batchSize, SizeType const seqLength)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        for (SizeType tokenIdx = tidx; tokenIdx < seqLength + 1; tokenIdx += blockDim.x * gridDim.x)
        {
            SizeType oldIndex = batchIdx * seqLength + tokenIdx;
            SizeType newIndex = batchIdx * (seqLength + 1) + tokenIdx;
            newMask[newIndex] = (tokenIdx < seqLength) ? oldMask[oldIndex] : 1;
        }
    }
}
} // namespace

void invokeExtendAttentionMask(ITensor& newMask, ITensor const& oldMask, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(TRTDataType<SizeType>::value == newMask.getDataType(), "attentionMask has wrong data type");
    TLLM_CHECK_WITH_INFO(TRTDataType<SizeType>::value == oldMask.getDataType(), "attentionMask has wrong data type");

    auto const& shape = oldMask.getShape();
    SizeType const batchSize = shape.d[0];
    SizeType const seqLength = shape.d[1];

    dim3 const blockSize(256, 1);
    dim3 const gridSize((seqLength + blockSize.x - 1) / blockSize.x, batchSize);

    extendAttentionMask<<<gridSize, blockSize, 0, stream.get()>>>(
        bufferCast<SizeType>(newMask), bufferCast<SizeType>(oldMask), batchSize, seqLength);
}

namespace
{
__global__ void copyInputToOutputTransposed(SizeType* outputIds, SizeType const* inputIds, SizeType const* inputLengths,
    SizeType const padId, SizeType const batchSize, SizeType const beamWidth, SizeType const maxInputLength)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const inputLength = inputLengths[batchIdx];
        for (SizeType tokenIdx = tidx; tokenIdx < maxInputLength; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const value = (tokenIdx < inputLength) ? inputIds[batchIdx * maxInputLength + tokenIdx] : padId;
            for (SizeType beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = tc::flat_index3(tokenIdx, batchIdx, beamIdx, batchSize, beamWidth);
                outputIds[outputIdx] = value;
            }
        }
    }
}
} // namespace

void invokeCopyInputToOutputTransposed(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputLengths,
    SizeType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        inputIds.getDataType() == outputIds.getDataType(), "Input and output have different data types");

    auto const batchSize = static_cast<SizeType>(inputLengths.getSize());
    auto const& inputShape = inputIds.getShape();
    SizeType const maxInputLength = inputShape.d[inputShape.nbDims - 1];
    auto const& outputShape = outputIds.getShape();
    SizeType const maxSeqLength = outputShape.d[0];
    SizeType const beamWidth = outputShape.d[2];

    auto const inputBatchSize = inputIds.getSize() / maxInputLength;
    TLLM_CHECK_WITH_INFO(std::size_t(batchSize) == inputBatchSize,
        common::fmtstr("Input ids batch size (%ld) does not match inputLengths size (%ld)", inputBatchSize,
            std::size_t(batchSize)));
    TLLM_CHECK_WITH_INFO(batchSize == outputShape.d[1],
        common::fmtstr(
            "Output ids batch size (%d) does not match inputLengths size (%d)", outputShape.d[1], batchSize));
    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "Output sequence length (%d) has to be larger than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxInputLength + blockSize.x - 1) / blockSize.x, batchSize);

    copyInputToOutputTransposed<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(outputIds),
        bufferCast<SizeType const>(inputIds), bufferCast<SizeType const>(inputLengths), padId, batchSize, beamWidth,
        maxInputLength);
}

namespace
{
__global__ void copyPackedInputToOutputTransposed(SizeType* outputIds, SizeType const* inputIds,
    SizeType const* inputOffsets, SizeType const padId, SizeType const batchSize, SizeType const beamWidth,
    SizeType const maxInputLength)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const tokenBegin = inputOffsets[batchIdx];
        auto const tokenEnd = inputOffsets[batchIdx + 1];
        auto const inputLength = tokenEnd - tokenBegin;

        for (SizeType tokenIdx = tidx; tokenIdx < maxInputLength; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const value = (tokenIdx < inputLength) ? inputIds[tokenBegin + tokenIdx] : padId;
            for (SizeType beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = tc::flat_index3(tokenIdx, batchIdx, beamIdx, batchSize, beamWidth);
                outputIds[outputIdx] = value;
            }
        }
    }
}
} // namespace

void invokeCopyPackedInputToOutputTransposed(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputOffsets,
    SizeType const maxInputLength, SizeType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        inputIds.getDataType() == outputIds.getDataType(), "Input and output have different data types");

    auto const batchSize = static_cast<SizeType>(inputOffsets.getSize()) - 1;
    auto const& outputShape = outputIds.getShape();
    SizeType const maxSeqLength = outputShape.d[0];
    SizeType const beamWidth = outputShape.d[2];

    TLLM_CHECK_WITH_INFO(batchSize == outputShape.d[1],
        common::fmtstr(
            "Output ids batch size (%d) does not match inputOffsets batch size (%d)", outputShape.d[1], batchSize));
    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "Output sequence length (%d) has to be larger than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxInputLength + blockSize.x - 1) / blockSize.x, batchSize);

    copyPackedInputToOutputTransposed<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(outputIds),
        bufferCast<SizeType const>(inputIds), bufferCast<SizeType const>(inputOffsets), padId, batchSize, beamWidth,
        maxInputLength);
}

namespace
{
__global__ void copyInputToOutput(SizeType* outputIds, SizeType const* inputIds, SizeType const* inputLengths,
    SizeType const padId, SizeType const batchSize, SizeType const beamWidth, SizeType const maxInputLength,
    SizeType const maxSeqLength)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const inputLength = inputLengths[batchIdx];
        for (SizeType tokenIdx = tidx; tokenIdx < maxInputLength; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const value = (tokenIdx < inputLength) ? inputIds[batchIdx * maxInputLength + tokenIdx] : padId;
            for (SizeType beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = tc::flat_index3(batchIdx, beamIdx, tokenIdx, beamWidth, maxSeqLength);
                outputIds[outputIdx] = value;
            }
        }
    }
}
} // namespace

void invokeCopyInputToOutput(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputLengths,
    SizeType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        inputIds.getDataType() == outputIds.getDataType(), "Input and output have different data types");

    auto const& inputShape = inputIds.getShape();
    auto const& outputShape = outputIds.getShape();
    TLLM_CHECK_WITH_INFO(
        outputShape.nbDims == 3, common::fmtstr("Output shape must have 3 dimensions, but has %d", outputShape.nbDims));

    auto const batchSize = static_cast<SizeType>(inputLengths.getSize());
    SizeType const maxInputLength = inputShape.d[inputShape.nbDims - 1];
    SizeType const beamWidth = outputShape.d[1];
    SizeType const maxSeqLength = outputShape.d[2];

    auto const inputBatchSize = inputIds.getSize() / maxInputLength;
    TLLM_CHECK_WITH_INFO(std::size_t(batchSize) == inputBatchSize,
        common::fmtstr("Input ids batch size (%ld) does not match inputLengths size (%ld)", inputBatchSize,
            std::size_t(batchSize)));
    TLLM_CHECK_WITH_INFO(batchSize == outputShape.d[0],
        common::fmtstr(
            "Output ids batch size (%d) does not match inputLengths size (%d)", outputShape.d[0], batchSize));
    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "Output sequence length (%d) has to be larger than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxInputLength + blockSize.x - 1) / blockSize.x, batchSize);

    copyInputToOutput<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(outputIds),
        bufferCast<SizeType const>(inputIds), bufferCast<SizeType const>(inputLengths), padId, batchSize, beamWidth,
        maxInputLength, maxSeqLength);
}

namespace
{
__global__ void copyPackedInputToOutput(SizeType* outputIds, SizeType const* inputIds, SizeType const* inputOffsets,
    SizeType const padId, SizeType const batchSize, SizeType const beamWidth, SizeType const maxInputLength,
    SizeType const maxSeqLength)
{
    SizeType const tidx = blockIdx.x * blockDim.x + threadIdx.x;
    SizeType const tidy = blockIdx.y * blockDim.y + threadIdx.y;

    for (SizeType batchIdx = tidy; batchIdx < batchSize; batchIdx += blockDim.y * gridDim.y)
    {
        auto const tokenBegin = inputOffsets[batchIdx];
        auto const tokenEnd = inputOffsets[batchIdx + 1];
        auto const inputLength = tokenEnd - tokenBegin;

        for (SizeType tokenIdx = tidx; tokenIdx < maxInputLength; tokenIdx += blockDim.x * gridDim.x)
        {
            auto const value = (tokenIdx < inputLength) ? inputIds[tokenBegin + tokenIdx] : padId;
            for (SizeType beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = tc::flat_index3(batchIdx, beamIdx, tokenIdx, beamWidth, maxSeqLength);
                outputIds[outputIdx] = value;
            }
        }
    }
}
} // namespace

void invokeCopyPackedInputToOutput(ITensor& outputIds, ITensor const& inputIds, ITensor const& inputOffsets,
    SizeType const maxInputLength, SizeType const padId, CudaStream const& stream)
{
    TLLM_CHECK_WITH_INFO(
        inputIds.getDataType() == outputIds.getDataType(), "Input and output have different data types");

    auto const& outputShape = outputIds.getShape();
    TLLM_CHECK_WITH_INFO(
        outputShape.nbDims == 3, common::fmtstr("Output shape must have 3 dimensions, but has %d", outputShape.nbDims));

    auto const batchSize = static_cast<SizeType>(inputOffsets.getSize()) - 1;
    SizeType const beamWidth = outputShape.d[1];
    SizeType const maxSeqLength = outputShape.d[2];

    TLLM_CHECK_WITH_INFO(batchSize == outputShape.d[0],
        common::fmtstr(
            "Output ids batch size (%d) does not match inputOffsets batch size (%d)", outputShape.d[0], batchSize));
    TLLM_CHECK_WITH_INFO(maxInputLength < maxSeqLength,
        common::fmtstr(
            "Output sequence length (%d) has to be larger than max input length (%d)", maxSeqLength, maxInputLength));

    dim3 const blockSize(256, 1);
    dim3 const gridSize((maxInputLength + blockSize.x - 1) / blockSize.x, batchSize);

    copyPackedInputToOutput<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<SizeType>(outputIds),
        bufferCast<SizeType const>(inputIds), bufferCast<SizeType const>(inputOffsets), padId, batchSize, beamWidth,
        maxInputLength, maxSeqLength);
}

namespace
{
template <typename T>
__global__ void scatterTensor(T* output, T const* input, std::uint32_t const batchSize,
    std::uint32_t const inputRowSize, std::size_t const outputRowSize, std::uint32_t const beamWidth)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const tidy = static_cast<std::size_t>(blockIdx.y) * blockDim.y + threadIdx.y;
    auto const stridex = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const stridey = static_cast<std::size_t>(blockDim.y) * gridDim.y;

    for (auto batchIdx = tidy; batchIdx < batchSize; batchIdx += stridey)
    {
        for (auto columnIdx = tidx; columnIdx < inputRowSize; columnIdx += stridex)
        {
            auto const inputIdx = batchIdx * inputRowSize + columnIdx;
            auto const value = input[inputIdx];
            std::size_t constexpr beamIdx{0};
            auto const outputIdx = (batchIdx * beamWidth + beamIdx) * outputRowSize + columnIdx;
            output[outputIdx] = value;
        }
    }
}

template <typename T>
__global__ void tileTensor(T* output, T const* input, std::uint32_t const batchSize, std::size_t const inputRowSize,
    std::size_t const outputRowSize, std::uint32_t const beamWidth)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const tidy = static_cast<std::size_t>(blockIdx.y) * blockDim.y + threadIdx.y;
    auto const stridex = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const stridey = static_cast<std::size_t>(blockDim.y) * gridDim.y;

    for (auto batchIdx = tidy; batchIdx < batchSize; batchIdx += stridey)
    {
        for (auto columnIdx = tidx; columnIdx < inputRowSize; columnIdx += stridex)
        {
            auto const inputIdx = batchIdx * inputRowSize + columnIdx;
            auto const value = input[inputIdx];
            for (std::size_t beamIdx = 0; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = (batchIdx * beamWidth + beamIdx) * outputRowSize + columnIdx;
                output[outputIdx] = value;
            }
        }
    }
}

template <typename T>
__global__ void tileTensorInPlace(
    T* inputOutput, std::uint32_t const batchSize, std::size_t const inputOutputRowSize, std::uint32_t const beamWidth)
{
    auto const tidx = static_cast<std::size_t>(blockIdx.x) * blockDim.x + threadIdx.x;
    auto const tidy = static_cast<std::size_t>(blockIdx.y) * blockDim.y + threadIdx.y;
    auto const stridex = static_cast<std::size_t>(blockDim.x) * gridDim.x;
    auto const stridey = static_cast<std::size_t>(blockDim.y) * gridDim.y;

    for (auto batchIdx = tidy; batchIdx < batchSize; batchIdx += stridey)
    {
        for (auto columnIdx = tidx; columnIdx < inputOutputRowSize; columnIdx += stridex)
        {
            auto const inputIdx = (batchIdx * beamWidth + 0) * inputOutputRowSize + columnIdx;
            auto const value = inputOutput[inputIdx];
            for (std::size_t beamIdx = 1; beamIdx < beamWidth; ++beamIdx)
            {
                auto const outputIdx = (batchIdx * beamWidth + beamIdx) * inputOutputRowSize + columnIdx;
                inputOutput[outputIdx] = value;
            }
        }
    }
}

} // namespace

template <typename T>
void invokeScatterTensor(ITensor& output, ITensor const& input, SizeType beamWidth, CudaStream const& stream)
{
    auto const& inputShape = input.getShape();
    auto const nbInputRows = static_cast<std::uint32_t>(inputShape.d[0]);
    auto const inputRowSize = input.getSize() / static_cast<std::size_t>(nbInputRows);
    auto const& outputShape = output.getShape();
    auto const nbOutputRows = static_cast<std::uint32_t>(outputShape.d[0]);
    auto const outputRowSize = output.getSize() / static_cast<std::size_t>(nbOutputRows);

    TLLM_CHECK_WITH_INFO(nbOutputRows == beamWidth * nbInputRows,
        common::fmtstr(
            "nbOutputRows (%d) must be beamWidth (%d) times nbInputRows (%d)", nbOutputRows, beamWidth, nbInputRows));
    TLLM_CHECK_WITH_INFO(outputRowSize >= inputRowSize,
        common::fmtstr("output row size (%ld) must be at least input row size (%ld)", outputRowSize, inputRowSize));

    dim3 const blockSize{256, 1};
    std::size_t const gridx{tc::ceilDiv(inputRowSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), nbInputRows};
    scatterTensor<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output), bufferCast<T const>(input),
        nbInputRows, inputRowSize, outputRowSize, static_cast<uint32_t>(beamWidth));
}

void scatterTensor(ITensor& output, ITensor const& input, SizeType beamWidth, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeScatterTensor<SizeType>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeScatterTensor<float>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kHALF: invokeScatterTensor<half>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kINT8: invokeScatterTensor<int8_t>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kFP8: invokeScatterTensor<__hip_fp8_e4m3_fnuz>(output, input, beamWidth, stream); break;
    default: TLLM_CHECK_WITH_INFO(false, "data type not supported");
    }
}

template <typename T>
void invokeTileTensor(ITensor& output, ITensor const& input, SizeType const beamWidth, CudaStream const& stream)
{
    auto const& inputShape = input.getShape();
    auto const nbInputRows = static_cast<std::uint32_t>(inputShape.d[0]);
    auto const inputRowSize = input.getSize() / static_cast<std::size_t>(nbInputRows);
    auto const& outputShape = output.getShape();
    auto const nbOutputRows = static_cast<std::uint32_t>(outputShape.d[0]);
    auto const outputRowSize = output.getSize() / static_cast<std::size_t>(nbOutputRows);

    TLLM_CHECK_WITH_INFO(nbOutputRows == beamWidth * nbInputRows,
        common::fmtstr(
            "nbOutputRows (%d) must be beamWidth (%d) times nbInputRows (%d)", nbOutputRows, beamWidth, nbInputRows));
    TLLM_CHECK_WITH_INFO(outputRowSize >= inputRowSize,
        common::fmtstr("output row size (%ld) must be at least input row size (%ld)", outputRowSize, inputRowSize));

    dim3 const blockSize{256, 1};
    std::size_t const gridx{tc::ceilDiv(inputRowSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), nbInputRows};
    tileTensor<<<gridSize, blockSize, 0, stream.get()>>>(bufferCast<T>(output), bufferCast<T const>(input), nbInputRows,
        inputRowSize, outputRowSize, static_cast<uint32_t>(beamWidth));
}

void tileTensor(ITensor& output, ITensor const& input, SizeType beamWidth, CudaStream const& stream)
{
    switch (input.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeTileTensor<SizeType>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeTileTensor<float>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kHALF: invokeTileTensor<half>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kINT8: invokeTileTensor<int8_t>(output, input, beamWidth, stream); break;
    case nvinfer1::DataType::kFP8: invokeTileTensor<__hip_fp8_e4m3_fnuz>(output, input, beamWidth, stream); break;
    default: TLLM_CHECK_WITH_INFO(false, "data type not supported");
    }
}

template <typename T>
void invokeTileTensorInPlace(ITensor& inputOutput, SizeType const beamWidth, CudaStream const& stream)
{
    auto const& inputOutputShape = inputOutput.getShape();
    auto const nbOutputRows = static_cast<std::uint32_t>(inputOutputShape.d[0]);
    auto const nbInputRows = nbOutputRows / static_cast<std::uint32_t>(beamWidth);
    auto const inputOutputRowSize = inputOutput.getSize() / static_cast<std::size_t>(nbOutputRows);

    dim3 const blockSize{256, 1};
    std::size_t const gridx{tc::ceilDiv(inputOutputRowSize, blockSize.x)};
    std::size_t const gridMax{std::numeric_limits<std::uint32_t>::max()};
    dim3 const gridSize{static_cast<std::uint32_t>(std::min(gridx, gridMax)), nbInputRows};
    tileTensorInPlace<<<gridSize, blockSize, 0, stream.get()>>>(
        bufferCast<T>(inputOutput), nbInputRows, inputOutputRowSize, static_cast<std::uint32_t>(beamWidth));
}

void tileTensorInplace(ITensor& tensor, SizeType beamWidth, CudaStream const& stream)
{
    switch (tensor.getDataType())
    {
    case nvinfer1::DataType::kINT32: invokeTileTensorInPlace<SizeType>(tensor, beamWidth, stream); break;
    case nvinfer1::DataType::kFLOAT: invokeTileTensorInPlace<float>(tensor, beamWidth, stream); break;
    case nvinfer1::DataType::kHALF: invokeTileTensorInPlace<half>(tensor, beamWidth, stream); break;
    case nvinfer1::DataType::kINT8: invokeTileTensorInPlace<int8_t>(tensor, beamWidth, stream); break;
    case nvinfer1::DataType::kFP8: invokeTileTensorInPlace<__hip_fp8_e4m3_fnuz>(tensor, beamWidth, stream); break;
    default: TLLM_CHECK_WITH_INFO(false, "data type not supported");
    }
}

} // namespace tensorrt_llm::runtime::kernels
