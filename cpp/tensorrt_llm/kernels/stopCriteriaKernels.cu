#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include "tensorrt_llm/common/cudaUtils.h"
#include "tensorrt_llm/common/reduceKernelUtils.cuh"
#include "tensorrt_llm/kernels/stopCriteriaKernels.h"

using namespace tensorrt_llm::common;

namespace tensorrt_llm
{
namespace kernels
{
__global__ void stopWordsCriterion(int32_t const** outputIds, int32_t const** parentIds, int32_t const** stopWords,
    FinishedState* finished, int32_t const* sequenceLengths, int32_t const* batchSlots, int32_t const* stopWordsLens,
    int32_t batchSize, int32_t beamWidth, int32_t maxSeqLen)
{
    int32_t const id = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t const batchIdx = blockIdx.y / beamWidth;
    int32_t const beamIdx = blockIdx.y % beamWidth;
    auto const batchSlot = batchSlots != nullptr ? batchSlots[batchIdx] : batchIdx;
    auto const batchBeamIdx = batchSlot * beamWidth + beamIdx;

    auto const* baseStopWords = stopWords[batchSlot];
    auto const stopWordsLen = stopWordsLens[batchSlot];
    auto const* baseOffsets = baseStopWords + stopWordsLen;

    if (id >= stopWordsLen || baseOffsets[id] < 0)
    {
        return;
    }

    auto const itemEnd = baseOffsets[id];
    auto const itemStart = (id > 0) ? baseOffsets[id - 1] : 0;
    auto const itemSize = itemEnd - itemStart;

    // The single-token case unconditionally bans the token
    bool shouldStop = false;

    // Need to minus 1 because the sequenceLengths is updated in this step
    auto const currentStep = sequenceLengths[batchBeamIdx] - 1;
    // Enough previously generated tokens to look for a match
    if (currentStep + 1 >= itemSize)
    {
        shouldStop = true;
        auto parentId = beamIdx;
        bool const gatherBeam = beamWidth > 1;

        for (int32_t tokenIdx = itemSize - 1; tokenIdx >= 0; tokenIdx--)
        {
            auto const previousToken
                = outputIds[batchSlot][parentId * maxSeqLen + currentStep - (itemSize - 1) + tokenIdx];
            if (previousToken != baseStopWords[itemStart + tokenIdx])
            {
                shouldStop = false;
                break;
            }
            if (gatherBeam)
            {
                parentId = parentIds == nullptr
                    ? 0
                    : parentIds[batchSlot][parentId * maxSeqLen + currentStep - (itemSize - 1) + tokenIdx];

                if (parentId < 0 || parentId >= beamWidth)
                {
                    shouldStop = false;
                    break;
                }
            }
        }
    }

    if (shouldStop)
    {
        finished[batchSlot * beamWidth + beamIdx].setFinishedStopWords();
    }
}

void invokeStopWordsCriterion(int32_t const** outputIds, int32_t const** parentIds, int32_t const** stopWords,
    FinishedState* finished, int32_t const* sequenceLengths, int32_t const* batchSlots, int32_t const* stopWordsLen,
    int32_t maxStopWordsLen, int32_t batchSize, int32_t beamWidth, int32_t maxSeqLen, hipStream_t stream)
{
    // Check if we have sampled a word from the stopWords list. If so, stop the sequence.
    dim3 block, grid;
    constexpr int32_t maxBlockSize{256};

    block.x = min(((maxStopWordsLen + 32 - 1) / 32) * 32, maxBlockSize);
    grid.x = (maxStopWordsLen + block.x - 1) / block.x;
    grid.y = batchSize * beamWidth;

    stopWordsCriterion<<<grid, block, 0, stream>>>(outputIds, parentIds, stopWords, finished, sequenceLengths,
        batchSlots, stopWordsLen, batchSize, beamWidth, maxSeqLen);
    sync_check_cuda_error();
}

__global__ void lengthCriterion(FinishedState* finished, int32_t* finishedSum, uint32_t const* sequenceLimitLength,
    int32_t const* sequenceLengths, int32_t const* batchSlots, int32_t batchSize, int32_t beamWidth)
{
    int32_t threadFinishedCount = 0;
    auto const batchIdx = blockIdx.x;
    auto const batchSlot = batchSlots != nullptr ? batchSlots[batchIdx] : batchIdx;

    for (int32_t beamIdx = threadIdx.x; beamIdx < beamWidth; beamIdx += blockDim.x)
    {
        auto const batchSlotBeamWidthIdx = batchSlot * beamWidth + beamIdx;

        auto finishState = finished[batchSlotBeamWidthIdx];

        if (sequenceLengths[batchSlotBeamWidthIdx] >= sequenceLimitLength[batchSlot])
        {
            finishState.setFinishedMaxLength();
        }
        threadFinishedCount += finishState.isFinished() ? 1 : 0;
        finished[batchSlotBeamWidthIdx] = finishState;
    }

    if (finishedSum)
    {
        int blockFinishedCount = 0;
        if (blockDim.x <= 32)
        {
            blockFinishedCount = warpReduceSum(threadFinishedCount);
        }
        else
        {
            blockFinishedCount = blockReduceSum(threadFinishedCount);
        }
        __syncthreads();

        if (threadIdx.x == 0)
        {
            finishedSum[batchSlot] = blockFinishedCount;
        }
    }
}

void invokeLengthCriterion(FinishedState* finished, int32_t* finishedSum, uint32_t const* sequenceLimitLength,
    int32_t const* sequenceLengths, int32_t const* batchSlots, int32_t batchSize, int32_t beamWidth,
    hipStream_t stream)
{
    // Check if we have attained the sequence length limit. If so, stop the
    // sequence. In addition, check if all sequences are stopped and return the
    // result in shouldStop
    dim3 block{min(512, uint32_t(beamWidth))};
    dim3 grid{uint32_t(batchSize)};

    lengthCriterion<<<grid, block, 0, stream>>>(
        finished, finishedSum, sequenceLimitLength, sequenceLengths, batchSlots, batchSize, beamWidth);
    sync_check_cuda_error();
}

} // namespace kernels
} // namespace tensorrt_llm
